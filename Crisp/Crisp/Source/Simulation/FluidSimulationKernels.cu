#include "hip/hip_runtime.h"
#include "FluidSimulationKernels.cuh"

#include <>

#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>

#include "Models/FluidSimulationParams.hpp"

namespace crisp
{
    namespace
    {
        __device__ __forceinline__ float poly6(float x)
        {
            if (x >= h)
                return 0.0f;

            float val = h2 - x * x;
            return poly6Const * val * val * val;
        }

        __device__ __forceinline__ glm::vec3 spikyGrad(glm::vec3 vec, float x)
        {
            return spikyGradConst * (h - x) * (h - x) * (1.0f / x) * vec;
        }

        __device__ __forceinline__ float viscoLaplacian(float x)
        {
            return viscosityLaplaceConst * (h - x);
        }

        __device__ __forceinline__ int3 calculateGridPosition(glm::vec4 particlePosition, float cellSize)
        {
            int3 gridPosition;
            gridPosition.x = static_cast<int>(particlePosition.x / cellSize);
            gridPosition.y = static_cast<int>(particlePosition.y / cellSize);
            gridPosition.z = static_cast<int>(particlePosition.z / cellSize);
            return gridPosition;
        }

        __device__ __forceinline__ int getGridLinearIndex(int3 gridPosition, const glm::ivec3& gridDims)
        {
            return gridPosition.z * gridDims.y * gridDims.x + gridPosition.y * gridDims.x + gridPosition.x;
        }

        __device__ __forceinline__ int getGridLinearIndex(int x, int y, int z, const glm::ivec3& gridDims)
        {
            return z * gridDims.y * gridDims.x + y * gridDims.x + x;
        }

        __device__ __forceinline__ int getGridLinearIndex(int x, int y, int z, const int3& gridDims)
        {
            return z * gridDims.y * gridDims.x + y * gridDims.x + x;
        }

        __device__ __forceinline__ glm::vec3 getHeatMapColor(float value) {
            if (value > 250.0f) {
                return glm::vec3(1.0f, 0.0f, 0.0f);
            }
            else if (value > 150) {
                return glm::mix(glm::vec3(1.0f, 1.0f, 0.0f), glm::vec3(1.0f, 0.0f, 0.0f), (value - 150) / 100.0f);
            }
            else if (value > 100) {
                return glm::mix(glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(1.0f, 1.0f, 0.0f), (value - 100) / 50.0f);
            }
            else if (value > 50) {
                return glm::mix(glm::vec3(0.0f, 1.0f, 1.0f), glm::vec3(0.0f, 1.0f, 0.0f), (value - 50) / 50.0f);
            }
            else
                return glm::mix(glm::vec3(0.0f, 0.0f, 1.0f), glm::vec3(0.0f, 1.0f, 1.0f), (value) / 50.0f);
        }
    }

    __global__ void computeGridIndex(int* gridIndices, int* particleIndices, const glm::vec4* positions, SimulationParams params)
    {
        int particleIdx = blockIdx.x * blockDim.x + threadIdx.x;
        if (particleIdx >= params.numParticles)
            return;
    
        glm::vec4 position = positions[particleIdx];
        int3 gridPosition = calculateGridPosition(position, params.cellSize);
        int linearGridIdx = getGridLinearIndex(gridPosition, params.gridDims);

        gridIndices[particleIdx]     = linearGridIdx;
        particleIndices[particleIdx] = particleIdx;
    }
    
    void FluidSimulationKernels::computeGridLocations(PropertyBuffer<int>& gridIndices, PropertyBuffer<int>& particleIndices, const PropertyBuffer<glm::vec4>& positions, const SimulationParams& params)
    {
        int numParticles = static_cast<int>(positions.getSize());
        int blockSize = 256;
        int gridSize = (numParticles - 1) / blockSize + 1;
        computeGridIndex<<<gridSize, blockSize>>>(gridIndices.getDeviceBuffer(), particleIndices.getDeviceBuffer(),
            positions.getDeviceBuffer(), params);
    }

    void FluidSimulationKernels::sortParticleIndicesByGridLocation(PropertyBuffer<int>& gridLocations, PropertyBuffer<int>& particleIndices)
    {
        auto gridIdxPtr     = thrust::device_pointer_cast(gridLocations.getDeviceBuffer());
        auto particleIdxPtr = thrust::device_pointer_cast(particleIndices.getDeviceBuffer());
        int numParticles    = static_cast<int>(particleIndices.getSize());
        thrust::sort_by_key(gridIdxPtr, gridIdxPtr + numParticles, particleIdxPtr);
    }

    void FluidSimulationKernels::computeGridCellOffsets(PropertyBuffer<int>& gridOffsets, const PropertyBuffer<int>& gridLocations, const PropertyBuffer<int>& gridIndices)
    {
        auto gridLocPtr        = thrust::device_pointer_cast(gridLocations.getDeviceBuffer());
        auto gridIdxPtr        = thrust::device_pointer_cast(gridIndices.getDeviceBuffer());
        auto gridCellOffsetPtr = thrust::device_pointer_cast(gridOffsets.getDeviceBuffer());
        int numCells = static_cast<int>(gridIndices.getSize());

        thrust::lower_bound(gridLocPtr, gridLocPtr + numCells,
                            gridIdxPtr, gridIdxPtr + numCells,
                            gridCellOffsetPtr);
    }

    __global__ void debugNbrs(glm::vec4* colors, const glm::vec4* positions, const int* gridOffsets, int numParticles, int3 gridDims, float cellSize)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        glm::vec4 position = positions[idx];
        int3 gridPosition = calculateGridPosition(position, cellSize);

        int3 lower;
        lower.x = max(0, gridPosition.x - 1);
        lower.y = max(0, gridPosition.y - 1);
        lower.z = max(0, gridPosition.z - 1);
        int3 upper;
        upper.x = min(gridDims.x - 1, gridPosition.x + 1);
        upper.y = min(gridDims.y - 1, gridPosition.y + 1);
        upper.z = min(gridDims.z - 1, gridPosition.z + 1);

        int numCells = gridDims.x * gridDims.y * gridDims.z;
        int count = 0;
        for (int z = lower.z; z <= upper.z; ++z)
        {
            for (int y = lower.y; y <= upper.y; ++y)
            {
                for (int x = lower.x; x <= upper.x; ++x)
                {
                    int linearCellIdx = getGridLinearIndex(x, y, z, gridDims);
                    int cellOffset = gridOffsets[linearCellIdx];
                    int nextCellOffset = linearCellIdx == numCells - 1 ? numParticles : gridOffsets[linearCellIdx + 1];
                    for (int i = cellOffset; i < nextCellOffset; ++i)
                    {
                        count++;
                    }
                }
            }
        }

        if (count == 64)
            colors[idx] = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
        else if (count == 3 * 3 * 2 * 8)
            colors[idx] = glm::vec4(0.0f, 1.0f, 1.0f, 0.0f);
        else
            colors[idx] = glm::vec4(0.0f, 1.0f, 0.0f, 1.0f);
    }

    void FluidSimulationKernels::debugNeighborCount(PropertyBuffer<glm::vec4>& colors, const PropertyBuffer<glm::vec4>& pos, const PropertyBuffer<int>& gridOffsets, int3 gridDims, float cellSize)
    {
        int n = static_cast<int>(colors.getSize());
        int blockSize = 256;
        int gridSize = (n - 1) / blockSize + 1;
        debugNbrs<<<gridSize, blockSize>>>(colors.getDeviceBuffer(), pos.getDeviceBuffer(), gridOffsets.getDeviceBuffer(), n, gridDims, cellSize);
    }

    __global__ void computeDensityAndPressureKernel(float* densities, float* pressures, glm::vec4* normals,
        const glm::vec4* positions, const int* gridOffsets, const int* gridLocations, const int* particleIndices,
        SimulationParams params)
    {
        int numParticles = params.numParticles;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        glm::vec4 position = positions[idx];
        int3 gridPosition = calculateGridPosition(position, params.cellSize);
        glm::ivec3 gridDims = params.gridDims;
        float poly6Const = params.poly6Const;
        

        int3 lower;
        lower.x = max(0, gridPosition.x - 1);
        lower.y = max(0, gridPosition.y - 1);
        lower.z = max(0, gridPosition.z - 1);
        int3 upper;
        upper.x = min(gridDims.x - 1, gridPosition.x + 1);
        upper.y = min(gridDims.y - 1, gridPosition.y + 1);
        upper.z = min(gridDims.z - 1, gridPosition.z + 1);

        int numCells = gridDims.x * gridDims.y * gridDims.z;
        glm::vec3 posI = position;
        float density = 0.0f;
        
        glm::vec3 ni;
        for (int z = lower.z; z <= upper.z; ++z)
        {
            for (int y = lower.y; y <= upper.y; ++y)
            {
                for (int x = lower.x; x <= upper.x; ++x)
                {
                    int linearCellIdx = getGridLinearIndex(x, y, z, gridDims);
                    int cellOffset = gridOffsets[linearCellIdx];
                    int nextCellOffset = linearCellIdx == numCells - 1 ? numParticles : gridOffsets[linearCellIdx + 1];
                    for (int i = cellOffset; i < nextCellOffset; ++i)
                    {
                        int j = particleIndices[i];
                        glm::vec3 posJ = positions[j];

                        glm::vec3 diff = posI - posJ;
                        float dist = glm::length(diff);

                        density += mass * poly6(dist);

                        //if (dist > 0.0f && dist < h)
                        //    ni += mass / densities[idx] * spikyGrad(diff, dist);
                    }
                }
            }
        }

        densities[idx] = density;
        pressures[idx] = max(0.0f, stiffness * (density - restDensity));
        normals[idx] = glm::vec4(h * ni, 1.0f);
        //pressures[idx] = stiffness * (density - restDensity);
    }

    void FluidSimulationKernels::computeDensityAndPressure(PropertyBuffer<float>& densities,           PropertyBuffer<float>& pressures, PropertyBuffer<glm::vec4>& normals,
                                            const PropertyBuffer<glm::vec4>& positions, const PropertyBuffer<int>& gridOffsets,
                                            const PropertyBuffer<int>& gridLocations,   const PropertyBuffer<int>& particleIndices,
                                            const SimulationParams& params)
    {
        int numParticles = static_cast<int>(positions.getSize());
        int blockSize = 256;
        int gridSize = (numParticles - 1) / blockSize + 1;
        computeDensityAndPressureKernel<<<gridSize, blockSize>>>(densities.getDeviceBuffer(), pressures.getDeviceBuffer(), normals.getDeviceBuffer(),
            positions.getDeviceBuffer(), gridOffsets.getDeviceBuffer(), gridLocations.getDeviceBuffer(), particleIndices.getDeviceBuffer(),
            params);
    }

    __global__ void computeForcesKernel(glm::vec3* forces, glm::vec4* colors,
        const glm::vec4* positions, const int* gridOffsets, const int* gridLocations, const int* particleIndices,
        const float* densities, const float* pressures, const glm::vec3* velocities, const glm::vec4* normals,
        int numParticles, int3 gridDims, float cellSize, const glm::vec3 gravity, float viscosity)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        glm::vec4 position = positions[idx];
        int3 gridPosition = calculateGridPosition(position, cellSize);


        int3 lower;
        lower.x = max(0, gridPosition.x - 1);
        lower.y = max(0, gridPosition.y - 1);
        lower.z = max(0, gridPosition.z - 1);
        int3 upper;
        upper.x = min(gridDims.x - 1, gridPosition.x + 1);
        upper.y = min(gridDims.y - 1, gridPosition.y + 1);
        upper.z = min(gridDims.z - 1, gridPosition.z + 1);

        int numCells = gridDims.x * gridDims.y * gridDims.z;
        glm::vec3 posI = position;
        glm::vec3 velI = velocities[idx];
        float pressureI = pressures[idx];
        float densityI  = densities[idx];

        glm::vec3 ni;
        int nbrs = 0;
        glm::vec3 fPressure(0.0f);
        glm::vec3 fVisco(0.0f);
        for (int z = lower.z; z <= upper.z; ++z)
        {
            for (int y = lower.y; y <= upper.y; ++y)
            {
                for (int x = lower.x; x <= upper.x; ++x)
                {
                    int linearCellIdx = getGridLinearIndex(x, y, z, gridDims);
                    int cellOffset = gridOffsets[linearCellIdx];
                    int nextCellOffset = linearCellIdx == numCells - 1 ? numParticles : gridOffsets[linearCellIdx + 1];
                    for (int i = cellOffset; i < nextCellOffset; ++i)
                    {
                        int j = particleIndices[i];
                        glm::vec3 posJ = positions[j];

                        glm::vec3 diff = posI - posJ;
                        float dist = glm::length(diff);

                        if (dist > 0.0f && dist < h)
                        {
                            float volJ = mass / densities[j];

                            fPressure += -volJ * (pressureI + pressures[j]) * 0.5f  * spikyGrad(diff, dist);
                            fVisco    +=  volJ * (velocities[j] - velI) * viscoLaplacian(dist);
                            ni += volJ * spikyGrad(diff, dist);
                        }
                        nbrs++;
                    }
                }
            }
        }
        
        glm::vec3 fGrav = densityI * gravity;

        glm::vec3 total = fPressure + viscosity * fVisco + fGrav;
        
        forces[idx] = total;

        glm::vec3 col = glm::normalize(total) * 0.5f + 0.5f;
        colors[idx] = glm::vec4(posI.y, 1.5f * posI.y, 1.0f, 1.0f);
        //colors[idx] = glm::vec4(glm::normalize(velI) * 0.5f + 0.5f, 1.0f);
        //colors[idx] = glm::vec4(glm::vec3(densityI / 1000.0f), 1);
        if (densityI < restDensity)
            colors[idx] = glm::vec4((restDensity - densityI) / 500.0f + 0.5f, 0.5f, 0.5f, 1.0f);
        else
            colors[idx] = glm::vec4(0.5f, 0.5f, (densityI - restDensity) / 500.0f + 0.5f, 1.0f);

        float norm = float(nbrs) / 255;
        //colors[idx] = glm::vec4(getHeatMapColor(float(nbrs)), 1.0f);
        colors[idx] = glm::vec4(glm::vec3(glm::length(ni * h)), 1.0f);
    }

    void FluidSimulationKernels::computeForces(PropertyBuffer<glm::vec3>& forces, PropertyBuffer<glm::vec4>& colors,
        const PropertyBuffer<glm::vec4>& positions, const PropertyBuffer<int>& gridOffsets, const PropertyBuffer<int>& gridLocations, const PropertyBuffer<int>& particleIndices,
        const PropertyBuffer<float>& densities, const PropertyBuffer<float>& pressures, const PropertyBuffer<glm::vec3>& velocities, const PropertyBuffer<glm::vec4>& normals,
        int3 gridDims, float cellSize, const glm::vec3& gravity, float viscosity)
    {
        int numParticles = static_cast<int>(positions.getSize());
        int blockSize = 256;
        int gridSize = (numParticles - 1) / blockSize + 1;
        computeForcesKernel<<<gridSize, blockSize>>>(forces.getDeviceBuffer(), colors.getDeviceBuffer(),
            positions.getDeviceBuffer(), gridOffsets.getDeviceBuffer(), gridLocations.getDeviceBuffer(), particleIndices.getDeviceBuffer(),
            densities.getDeviceBuffer(), pressures.getDeviceBuffer(), velocities.getDeviceBuffer(), normals.getDeviceBuffer(), numParticles, gridDims, cellSize, gravity, viscosity);
    }

    __global__ void integrateKernel(glm::vec4* positions, glm::vec3* velocities, glm::vec3* forces, float* densities, int numParticles, float3 fluidSpace, float timeStep)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        float step = timeStep;

        glm::vec3 a = forces[idx] / densities[idx];
        glm::vec3 velocity = velocities[idx] + step * a;

        glm::vec4 position = positions[idx] + glm::vec4(step * velocity, 0.0f);

        constexpr float damping = 0.5f;

        if (position.x < particleRadius)
        {
            position.x = particleRadius;
            velocity.x = damping * -velocity.x;
        }

        if (position.x > fluidSpace.x - particleRadius)
        {
            position.x = fluidSpace.x - particleRadius;
            velocity.x = damping * -velocity.x;
        }

        if (position.y < particleRadius)
        {
            position.y = particleRadius;
            velocity.y = damping * -velocity.y;
        }

        if (position.y > fluidSpace.y - particleRadius)
        {
            position.y = fluidSpace.y - particleRadius;
            velocity.y = damping * -velocity.y;
        }

        if (position.z < particleRadius)
        {
            position.z = particleRadius;
            velocity.z = damping * -velocity.z;
        }

        if (position.z > fluidSpace.z - particleRadius)
        {
            position.z = fluidSpace.z - particleRadius;
            velocity.z = damping * -velocity.z;
        }

        velocities[idx] = velocity;
        positions[idx]  = position;
    }

    void FluidSimulationKernels::integrate(PropertyBuffer<glm::vec4>& pos, PropertyBuffer<glm::vec3>& vel,
        const PropertyBuffer<glm::vec3>& forces, const PropertyBuffer<float>& densities, float3 fluidSpace, float timeStep)
    {
        int numParticles = static_cast<int>(pos.getSize());
        int blockSize = 256;
        int gridSize = (numParticles - 1) / blockSize + 1;
        integrateKernel<<<gridSize, blockSize>>>(pos.getDeviceBuffer(), vel.getDeviceBuffer(), forces.getDeviceBuffer(), densities.getDeviceBuffer(), numParticles, fluidSpace, timeStep);
    }
}